#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hip/hip_complex.h"

#define WARP_SIZE 32

// Kernel that executes on the CUDA device
template <int N>
__global__ void test(float *in, float *mul, float *out)
{
  int lane_id = threadIdx.x % WARP_SIZE;
  int warp_id = threadIdx.x / WARP_SIZE;

  __shared__ float smem [WARP_SIZE*2];
  volatile float* my_smem = &smem[WARP_SIZE*warp_id];
  __shared__ float smul[N];

  my_smem[lane_id] = in[lane_id];

  if (lane_id < N)
	smul[lane_id] = mul[lane_id];
  
  my_smem[lane_id] = smul[lane_id%N] * my_smem[lane_id];
  out[lane_id] = my_smem[lane_id];
}



// main routine that executes on the host
int main(void)
{
 
  const int N = 8;  // Number of elements in arrays
  size_t elems = 256;
  size_t size = elems * sizeof(float);
  
  float* in_h = (float*)calloc(elems, sizeof(float));        // Allocate array on host
  float* mul_h = (float*)calloc(N, sizeof(float));
  float* out_h = (float*)calloc(elems, sizeof(float));        // Allocate array on host

  for (int i = 0; i < elems; i++)
  {
    in_h[i] = (float)i;
  }
  for (int i = 0; i < N; i++)
  {
    mul_h[i] = 1.f;
  }
  
  float *in_d, *out_d, *mul_d;

  hipMalloc((void **) &in_d, size);   // Allocate array on device
  hipMalloc((void **) &out_d, size);   // Allocate array on device
  hipMalloc((void **) &mul_d, N*sizeof(float));   // Allocate array on device
  hipMemcpy(in_d, in_h, size, hipMemcpyDefault);
  hipMemcpy(out_d, out_h, size, hipMemcpyDefault);
  hipMemcpy(mul_d, mul_h, N*sizeof(float), hipMemcpyDefault);
  hipDeviceSynchronize();

  {    
    test<N> <<< 1, WARP_SIZE >>> (in_d, mul_d, out_d);
    hipDeviceSynchronize();
    hipMemcpy(out_h, out_d, WARP_SIZE*sizeof(float), hipMemcpyDefault);
    hipDeviceSynchronize();
  
    float expected = 0.0, got = 0.0, test = 0.0;
    for (int j = 0; j < 32; j++)
    {
      expected += in_h[j]*mul_h[j%N];
      got += out_h[j];
    }
   
    // Print results
    printf("sz: %d, Expected: %f, Got: %f\n", 32, expected, got);
  }  

  // Cleanup
  free(in_h);
  free(mul_h);
  free(out_h);
  hipFree(in_d);
  hipFree(out_d);
  hipFree(mul_d);

  return 0;
}
